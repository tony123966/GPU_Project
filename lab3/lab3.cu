#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include ""
#define R 0
#define G 1
#define B 2
__device__ __host__ int CeilDiv(int a, int b) { return (a - 1) / b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
	)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt + xt;
	if (yt < ht && xt < wt && mask[curt] > 127.0f) {
		const int yb = oy + yt, xb = ox + xt;
		const int curb = wb*yb + xb;
		if (0 <= yb && yb < hb && 0 <= xb && xb < wb) {
			output[curb * 3 + R] = target[curt * 3 + R];
			output[curb * 3 + G] = target[curt * 3 + G];
			output[curb * 3 + B] = target[curt * 3 + B];
		}
	}
}
__global__ void CalculateFixed(
	const float *background,
	const float *target,
	const float *mask,
	float *fixed,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
	)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt + xt;

	float targetNeiborSum = 0;
	float backgroundSum = 0;
	if (yt < ht && xt < wt) {
		const int yb = oy + yt, xb = ox + xt;
		const int curb = wb*yb + xb;
		
		for (int i = 0; i < 3; i++){
			targetNeiborSum=0;
			backgroundSum=0;
			/*
			targetNeiborLeft = (curt - 1 < 0) ? target[curt * 3 + i] : target[curt * 3 + i - 1 * 3];
			targetNeiborRight = (curt + 1 > wt*ht) ? target[curt * 3 + i] : target[curt * 3 + i + 1 * 3];
			targetNeiborUp = (curt - wt < 0) ? target[curt * 3 + i] : target[curt * 3 + i - wt * 3];
			targetNeiborDown = (curt + wt > wt*ht) ? target[curt * 3 + i] : target[curt * 3 + i + wt * 3];
					
			targetNeiborSum = targetNeiborLeft + targetNeiborRight + targetNeiborUp + targetNeiborDown;
			*/
			
			// ��
			if (curt - 1 < 0)
			{
				backgroundSum += background[curb * 3 + i - 1 * 3];
				targetNeiborSum += target[curt * 3 + i];//target��neibor�W�Xtarget��� target���[neibor��m �O�[�ۤv
			}
			else
			{
				backgroundSum += mask[curt - 1] > 127.0 ? 0 : background[curb * 3 + i - 1 * 3];
				targetNeiborSum += target[curt * 3 + i - 1 * 3];
			}
			// �k
			if (curt + 1 > wt*ht) 
			{
				backgroundSum += background[curb * 3 + i + 1 * 3];
				targetNeiborSum += target[curt * 3 + i];//target��neibor�W�Xtarget��� target���[neibor��m �O�[�ۤv
			}
			else
			{
				backgroundSum += mask[curt + 1] > 127.0 ? 0 : background[curb * 3 + i + 1 * 3];
				targetNeiborSum += target[curt * 3 + i + 1 * 3];
			}
			//�W
			if (curt - wt < 0)
			{
				backgroundSum += background[curb * 3 + i - wb * 3];
				targetNeiborSum += target[curt * 3 + i];//target��neibor�W�Xtarget��� target���[neibor��m �O�[�ۤv
			}
			else
			{
				backgroundSum += mask[curt - wt] > 127.0 ? 0 : background[curb * 3 + i - wb * 3];
				targetNeiborSum += target[curt * 3 + i - wt * 3];

			}
			//�U
			if (curt + wt > wt*ht)
			{
				backgroundSum += background[curb * 3 + i + wb * 3];
				targetNeiborSum += target[curt * 3 + i];//target��neibor�W�Xtarget��� target���[neibor��m �O�[�ۤv
			}
			else
			{
				backgroundSum += mask[curt + wt] > 127.0 ? 0 : background[curb * 3 + i + wb * 3];
				targetNeiborSum += target[curt * 3 + i + wt * 3];
			}
			/*
			targetNeiborSum = targetNeiborLeft + targetNeiborRight + targetNeiborUp + targetNeiborDown;
			backgroundSum = backgroundNeiborLeft + backgroundNeiborRight + backgroundNeiborUp + backgroundNeiborDown;*/
			fixed[curt * 3 + i] = 4.0 * target[curt * 3 + i] - targetNeiborSum + backgroundSum;
		}
	}

}
__global__ void PoissonImageCloningIteration(
	const float *fixed,
	const float *mask,
	float *target,
	float *output,
	const int wt, const int ht
	)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt + xt;

	float OutputNeiborSum=0;
	float w=1.8;//SOR paramater

	// w = 1 + (MAX_TIMES - time) / MAX_TIMES

	if (yt < ht && xt < wt && mask[curt] > 127.0f) {
		//Run 
		for (int i = 0; i < 3; i++){

			OutputNeiborSum = 0.0f;

			if (curt - 1 >= 0 && mask[curt - 1] > 127.0f) // ��
			{
				OutputNeiborSum += target[curt * 3 + i - 1 * 3];
			}
			if (curt + 1 < wt*ht && mask[curt + 1] > 127.0f) // �k
			{
				OutputNeiborSum += target[curt * 3 + i + 1 * 3];
			}
			if (curt - wt >= 0 && mask[curt - wt] > 127.0f) //�W
			{
				OutputNeiborSum += target[curt * 3 + i - wt * 3];
			}
			if (curt + wt < wt*ht && mask[curt + wt] > 127.0f) //�U
			{
				OutputNeiborSum += target[curt * 3 + i + wt * 3];
			}
			output[curt * 3 + i] = (1.0 / 4.0)*(fixed[curt * 3 + i] + OutputNeiborSum)*w + (1.0 - w)*output[curt * 3 + i];
		}
		
	}
}
void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
	)
{
	float *fixed, *buf1, *buf2;
	hipMalloc(&fixed, 3 * wt*ht*sizeof(float));
	hipMalloc(&buf1, 3 * wt*ht*sizeof(float));
	hipMalloc(&buf2, 3 * wt*ht*sizeof(float));

	dim3 gdim(CeilDiv(wt, 32), CeilDiv(ht, 16)), bdim(32, 16);

	CalculateFixed << <gdim, bdim >> >(
		background, target, mask, fixed,
		wb, hb, wt, ht, oy, ox
		);

	hipMemcpy(buf1, target, sizeof(float)* 3 * wt*ht, hipMemcpyDeviceToDevice);

	for (int i = 0; i < 5000; ++i)
	{
		PoissonImageCloningIteration << <gdim, bdim >> >(fixed, mask, buf1, buf2, wt, ht);
		PoissonImageCloningIteration << <gdim, bdim >> >(fixed, mask, buf2, buf1, wt, ht);
	}

	hipMemcpy(output, background, wb*hb*sizeof(float)* 3, hipMemcpyDeviceToDevice);


	SimpleClone << <gdim, bdim >> >(
		background, buf1, mask, output,
		wb, hb, wt, ht, oy, ox
		);

	/*SimpleClone << <dim3(CeilDiv(wt, 32), CeilDiv(ht, 16)), dim3(32, 16) >> >(
		background, target, mask, output,
		wb, hb, wt, ht, oy, ox
		);*/

	hipFree(fixed);
	hipFree(buf1);
	hipFree(buf2);
}
