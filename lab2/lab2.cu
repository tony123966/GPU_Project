#include "hip/hip_runtime.h"
﻿

#include "lab2.h"
static const unsigned W = 640;
static const unsigned H = 480;
static const unsigned NFRAME = 480;
#define bound(x) (x>255?255:x<0?0:x)
#define RGB2Y(R,G,B) bound((0.299*R)+(0.587*G)+(0.114*B))
#define RGB2U(R,G,B) bound((-0.169*R)+(-0.331*G)+(0.500*B)+128)
#define RGB2V(R,G,B) bound((0.500*R)+(-0.419*G)+(-0.081*B)+128)
#define M_PI  3.14
#define Two_M_PI  3.14*2
struct Lab2VideoGenerator::Impl {
	int t = 0;
};

Lab2VideoGenerator::Lab2VideoGenerator() : impl(new Impl) {
}

Lab2VideoGenerator::~Lab2VideoGenerator() {}

void Lab2VideoGenerator::get_info(Lab2VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};/*
  __device__ void Line(uint8_t *yuv, int x1, int y1, int x2, int y2, Vector3D color)
  {
  int idx = y1*W + x1;

  int rowOfY = (idx / W);
  int columnOfY = (idx%W);

  int rowOfUV = rowOfY / 2;
  int columnOfUV = columnOfY / 2;

  int uvWidth = W / 2;
  int diffX = abs(x1 - x2);
  int diffY = abs(y1 - y2);
  for (int i = 0; i<diffX; i++){
  for (int j = 0; j<diffY; j++){
  if ((rowOfY == ((y2 >= y1) ? (y1 + j) : (y1 - j))) && (columnOfY == ((x2 >= x1) ? (x1 + i) : (x1 - i)))){
  yuv[idx] = RGB2Y(color.x, color.y, color.z);
  yuv[W*H + rowOfUV *uvWidth + columnOfUV] = RGB2U(color.x, color.y, color.z);
  yuv[W*H + W*H / 4 + rowOfUV *uvWidth + columnOfUV] = RGB2V(color.x, color.y, color.z);
  }
  }
  }
  }
  __global__ void SetColor(uint8_t *yuv, Vector3D color)
  {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < W*H) {
  int rowOfY = (idx / W);
  int columnOfY = (idx%W);
  int rowOfUV = rowOfY/2;
  int columnOfUV = columnOfY / 2;
  int uvWidth=W/2;
  yuv[idx] = RGB2Y(0, 255, 0);
  yuv[W*H + rowOfUV *uvWidth + columnOfUV] = RGB2U(0, 255, 0);
  yuv[W*H + W*H / 4 + rowOfUV *uvWidth + columnOfUV] = RGB2V(0, 255, 0);
  //yuv[W*H + idx/4] = RGB2U(color.x, color.y, color.z);
  //yuv[W*H + W*H / 4 + idx/4] = RGB2V(color.x, color.y, color.z);
  }
  }
  __device__ void koch(uint8_t *yuv, int x1, int y1, int x2, int y2, int it, Vector3D colorRGB)
  {
  float angle = 60 * M_PI / 180;
  int x3 = (2 * x1 + x2) / 3;
  int y3 = (2 * y1 + y2) / 3;

  int x4 = (x1 + 2 * x2) / 3;
  int y4 = (y1 + 2 * y2) / 3;

  int x = x3 + (x4 - x3)*cos(angle) + (y4 - y3)*sin(angle);
  int y = y3 - (x4 - x3)*sin(angle) + (y4 - y3)*cos(angle);

  if (it > 0)
  {
  koch(yuv, x1, y1, x3, y3, it - 1, colorRGB);
  koch(yuv, x3, y3, x, y, it - 1, colorRGB);
  koch(yuv, x, y, x4, y4, it - 1, colorRGB);
  koch(yuv, x4, y4, x2, y2, it - 1, colorRGB);
  }
  else{
  Line(yuv, x1, y1, x3, y3, colorRGB);
  Line(yuv, x3, y3, x, y, colorRGB);
  Line(yuv, x, y, x4, y4, colorRGB);
  Line(yuv, x4, y4, x2, y2, colorRGB);
  }
  }

  __global__ void DrawLine(uint8_t *yuv, int x1, int y1, int x2, int y2, Vector3D color)
  {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < W*H){
  int rowOfY = (idx / W);
  int columnOfY = (idx%W);
  int rowOfUV = rowOfY / 2;
  int columnOfUV = columnOfY / 2;
  int uvWidth = W / 2;
  int diffX=abs(x1 - x2);
  int diffY = abs(y1 - y2);
  for (int i = 0; i<diffX; i++){
  for (int j = 0; j<diffY; j++){
  if ((rowOfY == ((y2>=y1) ? (y1 + j) : (y1 - j))) && (columnOfY ==(( x2>=x1) ? (x1 + i) : (x1 - i)))){
  yuv[idx] = RGB2Y(0, 255, 0);
  yuv[W*H + rowOfUV *uvWidth + columnOfUV] = RGB2U(0, 255, 0);
  yuv[W*H + W*H / 4 + rowOfUV *uvWidth + columnOfUV] = RGB2V(0, 255, 0);
  }
  }
  }

  //yuv[W*H + idx/4] = RGB2U(color.x, color.y, color.z);
  //yuv[W*H + W*H / 4 + idx/4] = RGB2V(color.x, color.y, color.z);
  }
  }
  __global__ void SetKoch(uint8_t *yuv, int t, Vector3D color)
  {
  int CenterX = W / 2;
  int CenterY = H / 2;
  int offset=20+t*2;
  int cloneNum=5;
  int iterNum = 6* t / NFRAME;
  int angle = Two_M_PI*t / NFRAME;
  for (int i = 0; i < cloneNum; i++){
  int newXPos = (CenterX + offset) * cosf(i * Two_M_PI / cloneNum) - (CenterY + offset) * sinf(i * Two_M_PI / cloneNum);
  int newYPos = (CenterX + +offset) * sinf(i *Two_M_PI / cloneNum) + (CenterY + offset) * cosf(i * Two_M_PI / cloneNum);
  koch(yuv, CenterX, CenterY, newXPos* cosf(angle) - newYPos*sinf(angle), newXPos*sinf(angle) + newYPos*cosf(angle), iterNum, color);
  }
  }

  void  Lab2VideoGenerator::CreateQuad(uint8_t *yuv, unsigned char size, int pos, Vector3D color)
  {
  //Y
  for (int i = 0; i< size; i++)
  {
  for (int j = 0; j < size; j++)
  {
  int pos_ = pos + (i * W + j)>W*H ? W*H : pos + (i * W + j);
  hipMemset(yuv + pos_, RGB2Y(color.x, color.y, color.z), 1);
  }
  }
  //U
  for (int i = 0; i< size / 2; i++)
  {
  for (int j = 0; j < size / 2; j++)
  {
  int pos_ = pos / 4 + (i * W / 2 + j)>W*H/4 ? W*H/4 : pos / 4 + (i * W / 2 + j);
  hipMemset(yuv + W*H + pos_, RGB2U(color.x, color.y, color.z), 1);
  }
  }
  //V
  for (int i = 0; i< size / 2; i++)
  {
  for (int j = 0; j < size / 2; j++)
  {
  int pos_ = pos / 4 + (i * W / 2 + j)>W*H / 4 ? W*H / 4 : pos / 4 + (i * W / 2 + j);
  hipMemset(yuv + W*H + W*H / 4 + pos_, RGB2V(color.x, color.y, color.z), 1);
  }
  }
  }
  void Lab2VideoGenerator::Generate(uint8_t *yuv) {//³Ð«Ø­Óframe
  //hipMemset(yuv, (impl->t)*255/NFRAME, W*H);
  //int posID = ((impl->t))*(W*H / (NFRAME));
  Vector3D background_ColorRGB(0, 0, 0);

  //background
  //Y
  hipMemset(yuv, RGB2Y(background_ColorRGB.x, background_ColorRGB.y, background_ColorRGB.z), W*H);
  //U
  hipMemset(yuv + W*H, RGB2U(background_ColorRGB.x, background_ColorRGB.y, background_ColorRGB.z), W*H / 4);
  //V
  hipMemset(yuv + W*H + W*H / 4, RGB2V(background_ColorRGB.x, background_ColorRGB.y, background_ColorRGB.z), W*H / 4);

  //CreateQuad(yuv, 64, posID, colorRGB);
  int block_dim = W*H / 10 + 1;
  //SetColor << <block_dim, 10 >> > (yuv,colorRGB);
  Vector3D colorRGB(0, 255, 0);
  SetKoch << <1, 1 >> > (yuv, (impl->t), colorRGB);
  ++(impl->t);
  }*/
void  Lab2VideoGenerator::CreateQuad(uint8_t *yuv, unsigned char size, int pos, Vector3D color)
{
	//Y
	for (int i = 0; i< size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			int pos_ = pos + (i * W + j)>W*H ? W*H : pos + (i * W + j);
			hipMemset(yuv + pos_, RGB2Y(color.x, color.y, color.z), 1);
		}
	}
	//U
	for (int i = 0; i< size / 2; i++)
	{
		for (int j = 0; j < size / 2; j++)
		{
			int pos_ = pos / 4 + (i * W / 2 + j)>W*H / 4 ? W*H / 4 : pos / 4 + (i * W / 2 + j);
			hipMemset(yuv + W*H + pos_, RGB2U(color.x, color.y, color.z), 1);
		}
	}
	//V
	for (int i = 0; i< size / 2; i++)
	{
		for (int j = 0; j < size / 2; j++)
		{
			int pos_ = pos / 4 + (i * W / 2 + j)>W*H / 4 ? W*H / 4 : pos / 4 + (i * W / 2 + j);
			hipMemset(yuv + W*H + W*H / 4 + pos_, RGB2V(color.x, color.y, color.z), 1);
		}
	}
}
__global__ void SetColor(uint8_t *yuv, Vector3D color)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < W*H) {
		int rowOfY = (idx / W);
		int columnOfY = (idx%W);
		int rowOfUV = rowOfY / 2;
		int columnOfUV = columnOfY / 2;
		int uvWidth = W / 2;
		yuv[idx] = RGB2Y(0, 255, 0);
		yuv[W*H + rowOfUV *uvWidth + columnOfUV] = RGB2U(0, 255, 0);
		yuv[W*H + W*H / 4 + rowOfUV *uvWidth + columnOfUV] = RGB2V(0, 255, 0);
		//yuv[W*H + idx/4] = RGB2U(color.x, color.y, color.z);
		//yuv[W*H + W*H / 4 + idx/4] = RGB2V(color.x, color.y, color.z);
	}
}
__device__ void Line(uint8_t *yuv, int x1, int y1, int x2, int y2, Vector3D color)
{
	unsigned char colorZ[3] = { 0, 0, 255 };
	int windowDis = W;
	int dis = x1;
	float disMod = (float)dis / (float)windowDis;

	int idx = y1*W + x1;
	if (idx>W*H || x1<0 || x2<0 || x1>W || x2>W || y1<0 || y2<0 || y1>H || y2>H)return;
	//printf("%f \n", disMod);
	int rowOfY = (idx / W);
	int columnOfY = (idx%W);
	int rowOfUV = rowOfY / 2;
	int columnOfUV = columnOfY / 2;
	int uvWidth = W / 2;
	int uvIdx = rowOfUV *uvWidth + columnOfUV; /*
	yuv[idx] = RGB2Y(color.x*disMod + colorZ[0] * (1 - disMod), 
	                 color.y*disMod + colorZ[1] * (1 - disMod), 
	                  color.z*disMod+ colorZ[2] * (1 - disMod));
	//printf("x=%f,y=%f,z=%f\n", color.x*disMod + colorZ[0] * (1 - disMod), color.y*disMod + colorZ[1] * (1 - disMod), color.z*disMod + colorZ[2] * (1 - disMod));
	yuv[W*H + uvIdx] = RGB2U(color.x*disMod + colorZ[0] * (1 - disMod), 
							 color.y*disMod + colorZ[1] * (1 - disMod), 
							 color.z*disMod + colorZ[2] * (1 - disMod));
	yuv[W*H + W*H / 4 + uvIdx] = RGB2V(color.x*disMod + colorZ[0] * (1 - disMod), 
	                                    color.y*disMod + colorZ[1] * (1 - disMod),
	                                    color.z*disMod + colorZ[2] * (1 - disMod));*/
	yuv[idx] = RGB2Y(color.x, abs(color.y - disMod * color.y), abs(color.z - disMod * color.z));
	yuv[W*H + uvIdx] = RGB2U(color.x, abs(color.y - disMod *color.y), abs(color.z - disMod * color.z));
	yuv[W*H + W*H / 4 + uvIdx] = RGB2V(color.x, abs(color.y - disMod * color.y), abs(color.z - disMod * 95));
}

__device__ void koch(uint8_t *yuv, int x1, int y1, int x2, int y2, int it, Vector3D colorRGB)
{
	float angle = 60 * M_PI / 180;
	int x3 = (2 * x1 + x2) / 3;
	int y3 = (2 * y1 + y2) / 3;

	int x4 = (x1 + 2 * x2) / 3;
	int y4 = (y1 + 2 * y2) / 3;

	int x = x3 + (x4 - x3)*cos(angle) + (y4 - y3)*sin(angle);
	int y = y3 - (x4 - x3)*sin(angle) + (y4 - y3)*cos(angle);

	if (it > 0)
	{
		koch(yuv, x1, y1, x3, y3, it - 1, colorRGB);
		koch(yuv, x3, y3, x, y, it - 1, colorRGB);
		koch(yuv, x, y, x4, y4, it - 1, colorRGB);
		koch(yuv, x4, y4, x2, y2, it - 1, colorRGB);
	}
	else{
		Line(yuv, x1, y1, x3, y3, colorRGB);
		Line(yuv, x3, y3, x, y, colorRGB);
		Line(yuv, x, y, x4, y4, colorRGB);
		Line(yuv, x4, y4, x2, y2, colorRGB);
	}
}
__global__ void SetKoch(uint8_t *yuv, int x1, int y1, int  offset, int it, int cloneNum, float angle, Vector3D colorRGB)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < cloneNum)
	{
		//printf("it=%d ,ff=%d \n", it, offset);
		idx += 1;
		float param = (360.0 / cloneNum)*idx;
		int newXPos = (float)(x1 + offset) * cosf(param*Two_M_PI / 360.0) - (float)(y1)* sinf(param*Two_M_PI / 360.0);
		int newYPos = (float)(x1 + offset) * sinf(param*Two_M_PI / 360.0) + (float)(y1)* cosf(param*Two_M_PI / 360.0);
		koch(yuv, x1, y1, newXPos*cosf(angle) - newYPos*sinf(angle), newYPos*cosf(angle) + newXPos*sinf(angle), it, colorRGB);
	}
}
void Lab2VideoGenerator::Generate(uint8_t *yuv) {//³Ð«Ø­Óframe
	//hipMemset(yuv, (impl->t)*255/NFRAME, W*H);
	int posID = ((impl->t))*(W*H / (NFRAME));
	Vector3D background_ColorRGB(255,255, 250);

	//background
	//Y
	hipMemset(yuv, RGB2Y(background_ColorRGB.x, background_ColorRGB.y, background_ColorRGB.z), W*H);
	//U
	hipMemset(yuv + W*H, RGB2U(background_ColorRGB.x, background_ColorRGB.y, background_ColorRGB.z), W*H / 4);
	//V
	hipMemset(yuv + W*H + W*H / 4, RGB2V(background_ColorRGB.x, background_ColorRGB.y, background_ColorRGB.z), W*H / 4);


	int block_dim = W*H / 10 + 1;
	//SetColor << <block_dim, 10 >> > (yuv,colorRGB);

	Vector3D colorRGBX(255, 0, 0);
	Vector3D colorRGBY(0, 255,0);
	Vector3D colorRGBZ(0, 0, 255);
	int CenterX = W / 2;
	int CenterY = H / 2;

	int cloneNum = 10;
	int offset = 20 + impl->t * 2;
	int iterNum = 5 * impl->t / NFRAME;
	float angle = (float)Two_M_PI*(impl->t + 1) /(NFRAME);
	//CreateQuad(yuv, 8, posID, colorRGBY);
	SetKoch << <1, cloneNum >> > (yuv, CenterX, CenterY, offset, iterNum, cloneNum, angle, colorRGBX);
	SetKoch << <1, cloneNum >> > (yuv, CenterX, CenterY, offset, iterNum, cloneNum, -angle, colorRGBY);
	//SetKochZ << <1,1 >> > (yuv, CenterX, CenterY, 100, iterNum, colorRGBZ);
	(impl->t)++;
}
