#include "hip/hip_runtime.h"
#include <random>
#include <vector>
#include <tuple>
#include <cstdio>
#include <cstdlib>
#include <functional>
#include <algorithm>
#include "SyncedMemory.h"
#include "Timer.h"
#include "counting.h"
using namespace std;

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

template <typename Engine>
tuple<vector<char>, vector<int>, vector<int>> GenerateTestCase(Engine &eng, const int N) {
	srand((unsigned)time(NULL));
	poisson_distribution<int> pd(14.0);
	bernoulli_distribution bd(0.1);
	uniform_int_distribution<int> id1(1, 20);
	uniform_int_distribution<int> id2(1, 5);
	uniform_int_distribution<int> id3('a', 'z');
	tuple<vector<char>, vector<int>, vector<int>> ret;
	auto &text = get<0>(ret);
	auto &pos = get<1>(ret);
	auto &head = get<2>(ret);
	auto gen_rand_word_len = [&] () -> int {
		return max(1, min(500, pd(eng) - 5 + (bd(eng) ? id1(eng)*20 : 0)));
	};
	auto gen_rand_space_len = [&] () -> int {
		return id2(eng);
	};
	auto gen_rand_char = [&] () {
		return id3(eng);
	};
	auto AddWord = [&] () {
		head.push_back(text.size());
		int n = gen_rand_word_len();
		for (int i = 0; i < n; ++i) {
			text.push_back(gen_rand_char());
			//printf("%c", text[i]);
			pos.push_back(i+1);
		}
	};
	auto AddSpace = [&] () {
		int n = gen_rand_space_len();
		for (int i = 0; i < n; ++i) {
			text.push_back('\n');
			//printf("\n");
			pos.push_back(0);
		}
	};

	AddWord();
	while (text.size() < N) {
		AddSpace();
		AddWord();
	}

	/*int i=0;
	while (i<text.size()) {
		printf("%c", text[i]);
		i++;
	}
	printf("\n");*/
	
	return ret;
}

int main(int argc, char **argv)
{

	// Initialize random text
	default_random_engine engine(12345);
	auto text_pos_head = GenerateTestCase(engine, 512); // 40 MB data
	vector<char> &text = get<0>(text_pos_head);
	vector<int> &pos = get<1>(text_pos_head);
	vector<int> &head = get<2>(text_pos_head);


	// Prepare buffers
	int n = text.size();
	char *text_gpu;
	hipMalloc(&text_gpu, sizeof(char)*n);//�N�Ҧ���r�Ǩ�text_gpu
	SyncedMemory<char> text_sync(text.data(), text_gpu, n);
	text_sync.get_cpu_wo(); // touch the cpu data
	MemoryBuffer<int> pos_yours(n), head_yours(n);
	auto pos_yours_sync = pos_yours.CreateSync(n);
	auto head_yours_sync = head_yours.CreateSync(n);

	// Create timers
	Timer timer_count_position;

	// Part I
	timer_count_position.Start();
	int *pos_yours_gpu = pos_yours_sync.get_gpu_wo();
	hipMemset(pos_yours_gpu, 0, sizeof(int)*n);
	CountPosition(text_sync.get_gpu_ro(), pos_yours_gpu, n);

	CHECK;
	timer_count_position.Pause();
	printf_timer(timer_count_position);
	// Part I check
	const int *golden = pos.data();
	const int *yours = pos_yours_sync.get_cpu_ro();
	int n_match1 = mismatch(golden, golden+n, yours).first - golden;
	printf("%d\n",n);
	/*for (int i = 0; i<n; i++)
	{
		printf("%d=%d\n", i, pos_yours_sync.get_cpu_ro()[i]);
	}*/
	printf("%d\n", n_match1);
	if (n_match1 != n) {
		puts("Part I WA!");
		copy_n(golden, n, pos_yours_sync.get_cpu_wo());
	}

	// Part II
	int *head_yours_gpu = head_yours_sync.get_gpu_wo();
	hipMemset(head_yours_gpu, 0, sizeof(int)*n);
	int n_head = ExtractHead(pos_yours_sync.get_gpu_ro(), head_yours_gpu, n);
	CHECK;
	printf("%d__\n", n_head);
	// Part II check
	do {
		if (n_head != head.size()) {
			n_head = head.size();
			puts("Part II WA (wrong number of heads)!");
		} else {
			int n_match2 = mismatch(head.begin(), head.end(), head_yours_sync.get_cpu_ro()).first - head.begin();
			if (n_match2 != n_head) {
				puts("Part II WA (wrong heads)!");
			} else {
				break;
			}
		}
		copy_n(head.begin(), n_head, head_yours_sync.get_cpu_wo());
	} while(false);
	/*for (int i = 0; i<n_head; i++)
	{
		printf("%d=%d\n", i, head_yours_sync.get_cpu_ro()[i]);
	}*/
	// Part III
	// Do whatever your want
	Part3(text_gpu, pos_yours_sync.get_gpu_rw(), head_yours_sync.get_gpu_rw(), n, n_head);
	CHECK;
	/*for (int i = 0; i<n; i++)
	{
		printf("%d=%d\n", i, pos_yours_sync.get_cpu_ro()[i]);
	}*/
	hipFree(text_gpu);
	system("pause");
	return 0;
}
class NumArray {
	vector<int> BITreeArray;
	vector<int> orgNums;
public:
	NumArray(vector<int> &nums) {
		int size = nums.size();
		if (size == 0) return;

		BITreeArray = vector<int>(nums.size() + 1, 0);
		orgNums = vector<int>(nums.size(), 0);

		//store the actual values in BITree
		for (int i = 0; i < size; i++)
		{
			update(i, nums[i]);
		}
		orgNums = nums;
	}

	void update(int i, int val) {
		int idx = i + 1; //idx in BITree is 1 more than the index in arr
		int diff = val - orgNums[i];
		orgNums[i] = val;

		while (idx <= orgNums.size())
		{
			BITreeArray[idx] += diff;
			idx += idx & (-idx);
		}
	}
	int getSum(int j)
	{
		int sum = 0;
		int idx = j + 1;//convert to BITree index

		while (idx >0)
		{
			sum += BITreeArray[idx];
			idx -= idx & (-idx);
		}
		return sum;
	}
	int sumRange(int i, int j) {

		if (i == 0)
			return getSum(j);
		else
			return getSum(j) - getSum(i - 1);
	}
};