#include "hip/hip_runtime.h"
﻿#include "counting.h"
#include "SyncedMemory.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include <cstdlib>
#include <iostream>
#include <vector>
#include <math.h>
using namespace std;
__device__ __host__ int CeilDiv(int a, int b)
{
	return (a - 1) / b + 1;
}
__device__ __host__ int CeilAlign(int a, int b)
{
	return CeilDiv(a, b) * b;
}

__constant__ char* text;
__global__ void EstablishButtonTree(const char* text, int* pos, int dataSize) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("tree :%d %d\n", idx, dataSize);
	if (idx < dataSize) {
		if (text[idx] == '\n') pos[idx] = 0;
		else pos[idx] = 1;
		//printf("i=%d,idx = %d, %d\n", dataSize,idx, pos[idx]);
	}
}
__global__ void EstablishLayerTree(int* layer_prev, int* layer, int layer_size_prev) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("tree :%d %d\n", idx, layer_size);
	if (idx < layer_size_prev && idx % 2 == 0) {
		if (layer_prev[idx] & layer_prev[idx + 1])
			layer[idx/2] = layer_prev[idx] + layer_prev[idx+1];
		else
			layer[idx/2] = 0;
	}
	//printf("i=%d idx = %d, %d\n", layer_size_prev/2, idx, layer[idx]);
}
__device__ int xyToIndex(int y, int x, int text_size)
{
	int result = 0;
	for (int i = 0; i<y; i++) {
		result += (int)(text_size * pow(0.5, i));
	}
	result += x;
	return result;
}
__device__ int SetPositionNonRecursive(int curIdx, int *treeLayerBeginSize, int text_size)
{
	int curTreeIdx=0;
	int index=curIdx;
	int countNum=0;
	bool isTopDown=false;
	while (1){
		if (curIdx % 2 == 1 && !isTopDown)//右葉節點
		{
			if (treeLayerBeginSize[xyToIndex(curTreeIdx + 1, curIdx / 2, text_size)] == 0 && curTreeIdx == 0) { //若父親是0
				countNum+=1;
				return countNum;
			}
			if (curTreeIdx != 0)
			{
				while (1)//往父親走
				{
					curIdx /= 2; curTreeIdx += 1;
					if (curIdx % 2 == 0)//若父親是左節點
					{
						countNum += treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx, text_size)];
						if (!curIdx) return countNum;
						if (treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx - 1, text_size)] == 0) { // 開始topDown
							isTopDown=true;
							break;
						}
						else//向左走
						{
							curIdx-=1;
						}
					}
					if (curIdx % 2 ==1)//若父親是右節點
					{
						if (treeLayerBeginSize[xyToIndex(curTreeIdx + 1, curIdx / 2, text_size)] == 0)
						{
							countNum += treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx, text_size)];
							curIdx -= 1;
							// 開始topDown
							isTopDown=true;
							break;
						}
					}
					if (isTopDown)break;
				}
			}
			if (isTopDown)break;
		}
		else if (curIdx % 2 == 0 && !isTopDown)//左葉節點
		{
			countNum += treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx, text_size)];
			if (!curIdx) return countNum;
			if (curTreeIdx == 0 && treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx - 1, text_size)] == 0)return countNum;
			if (treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx - 1, text_size)] != 0) {
				curIdx--;
			}

		}
		if (isTopDown)break;
	}
	if (isTopDown){
		while (1)
		{
			if (treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx, text_size)] == 0 && curIdx % 2 == 0){//遇到0了且是左子點
				if (curTreeIdx == 0) return countNum;
				curTreeIdx += 1, curIdx / 2;
			}
			else if (curIdx % 2 == 1){ // right
				if (treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx, text_size)] != 0){
					countNum += treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx, text_size)];
					curIdx -=1;
				}
				else
				{
					if (curTreeIdx == 0) return countNum;
					curTreeIdx += 1, curIdx / 2;
				}
			}
		}
	}
	return countNum;
}
__device__ int SetPositionRecursive(int curTreeIdx, int curIdx, int *treeLayerBeginSize, bool isTopDown, int countNum, int text_size)
{
	if (!isTopDown)
	{
		if (treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx, text_size)])
		{
			if (curTreeIdx == 0)//底層
			{
				if (curIdx % 2 == 0){//左子點
					countNum += 1;
					if (!curIdx) return countNum;
					if (treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx-1, text_size)] != 0) {
						return SetPositionRecursive(curTreeIdx, curIdx - 1, treeLayerBeginSize, false, countNum, text_size);
					}
					return countNum;
				}
				else if (curIdx % 2 == 1){// 右子點
					if (treeLayerBeginSize[xyToIndex(curTreeIdx + 1, curIdx / 2, text_size)] != 0) return SetPositionRecursive(curTreeIdx + 1, curIdx / 2, treeLayerBeginSize, false, countNum, text_size);
					else {
						if (treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx, text_size)]) {
							countNum++;
							//return currentNum;
						}
						return countNum;
					}
				}	
			}
			else//非底層
			{
				if (curIdx % 2 == 0)//左子點
				{
					countNum += treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx, text_size)];
					if (!curIdx) return countNum;
					if (treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx-1, text_size)] == 0) { // topDown
						return SetPositionRecursive(curTreeIdx, curIdx - 1, treeLayerBeginSize, true, countNum, text_size);
					}
					return	SetPositionRecursive(curTreeIdx, curIdx - 1, treeLayerBeginSize, false, countNum, text_size);
				}
				else if(curIdx % 2 == 1)// 右子點
				{
					if (treeLayerBeginSize[xyToIndex(curTreeIdx+1, curIdx/2, text_size)] != 0)
					{
						return SetPositionRecursive(curTreeIdx + 1, curIdx / 2, treeLayerBeginSize, false, countNum, text_size);
					}
					else{
						countNum += treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx, text_size)];
						return SetPositionRecursive(curTreeIdx, curIdx - 1, treeLayerBeginSize, true, countNum, text_size);
					}
				}
			}	
		}
	}//topDown
	else{
		if (treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx, text_size)] == 0 && curIdx % 2 == 0){//遇到0了且是左子點
			if (curTreeIdx == 0) return countNum;
			return SetPositionRecursive(curTreeIdx - 1, curIdx * 2 + 1, treeLayerBeginSize, true, countNum,text_size);
		}
		else if (curIdx % 2 == 1){ // right
			if (treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx, text_size)] != 0){
				countNum += treeLayerBeginSize[xyToIndex(curTreeIdx, curIdx, text_size)];
				return SetPositionRecursive(curTreeIdx, curIdx - 1, treeLayerBeginSize, true, countNum, text_size);
			}
			else
			{
				if (curTreeIdx == 0) return countNum;
				return SetPositionRecursive(curTreeIdx - 1, curIdx * 2 + 1, treeLayerBeginSize, true, countNum, text_size);
			}
		}
	}
	return countNum;
}
__global__ void SetPosition(int  *allTree, int * treeResult, int text_size)
{
	/*
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("%d %d\n", idx, text_size);
	if (idx < text_size) {
		printf("ttt");
		int curIdx = idx;
		int height = 0;
		int count = 0;
		if (curIdx % 2 == 1)//右葉節點
		{
			if (curIdx / 2 >= text_size / 2)return;
			if (allTree[xyToIndex(height + 1, curIdx / 2, text_size)] == 0) { //若父親是0
				return;
			}
			else//若父親不是0
			{
				for (; height + 1 <= 9;)//不停往父親走
				{
					if (curIdx / 2 >= text_size / powf(2, height + 1))return;
					curIdx /= 2; height += 1;
					if (curIdx % 2 == 0)//若父親是左節點
					{
						if (curIdx - 1 < 0)//沒有左邊
						{
							return;
						}
						else//有左邊
						{
							if (allTree[xyToIndex(height, curIdx-1, text_size)] == 0)//左邊的是0
							{
								//開始topDown
								count += allTree[xyToIndex(height, curIdx, text_size)];
								curIdx -= 1;//往左邊平移
								for (; height - 1 > 0;)
								{
									curIdx = 2 * curIdx + 1; height -= 1;
									if (allTree[xyToIndex(height, curIdx, text_size)] != 0)
									{
										count += allTree[xyToIndex(height, curIdx, text_size)];
										allTree[xyToIndex(height, curIdx, text_size)] = count;
										return;
									}
								}
							}
							else//左邊的不是0
							{
								count += allTree[xyToIndex(height, curIdx, text_size)];
								curIdx -= 1;//往左邊平移
							}
						}
					}
					else//父親是右節點
					{
						if (allTree[xyToIndex(height, curIdx, text_size)] == 0)//找到0為止
						{
							curIdx = 2 * curIdx + 1;//走回子節點
							height -= 1;
							//開始topDown
							count += allTree[xyToIndex(height, curIdx, text_size)];
							if (curIdx - 1 < 0)//沒有左邊
							{
								return;
							}
							else{//有左邊
								curIdx -= 1;//往左邊平移
								for (; height - 1 > 0;)
								{
									curIdx = 2 * curIdx + 1; height -= 1;
									if (allTree[xyToIndex(height, curIdx, text_size)] != 0)
									{
										count += allTree[xyToIndex(height, curIdx, text_size)];
										allTree[xyToIndex(height, curIdx, text_size)] = count;
										return;
									}
								}
							}
						}
					}
				}
			}

		}
		else//左葉節點
		{
		}
	}
	*/
	
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < text_size)
	{
		treeResult[idx] = SetPositionRecursive(0, idx, allTree, false, 0, text_size);
		//treeResult[idx] = SetPositionNonRecursive(idx, allTree, text_size);
	}
}
__global__ void SetCount(int *pos, int  text_size)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int count = 1;
	if (idx<text_size && pos[idx] == 1)
	{
		int index = idx;
		count = 1;
		while (index - 1 >= 0)
		{
			if (pos[index - 1]>0){
				count++;
				index -= 1;
			}
			else break;
		}
		pos[idx] = count;
	}
}
void CountPosition(const char *text, int *pos, int text_size)
{
	int * treeArray_gpu;
	int *treeLayer[10];

	for (int i = 0, size = text_size; i< 10; i++, size = size / 2){
		treeLayer[i] = (int *)malloc(sizeof(int)*size);
	}

	hipMalloc((void**)& treeArray_gpu, sizeof(int)* text_size);
	hipMemcpy(treeArray_gpu, treeLayer[0], sizeof(int)*text_size, hipMemcpyHostToDevice);
	//底層
	int block_dim = text_size / 512 + 1;
	EstablishButtonTree << <block_dim, 512 >> >(text, treeArray_gpu, text_size);//想要用GPU計算一定要先hipMalloc一段GPU記憶體空間
	hipDeviceSynchronize();
	hipMemcpy(treeLayer[0], treeArray_gpu, sizeof(int)*text_size, hipMemcpyDeviceToHost);//將GPU算完的結果拿回CPU

	int totalSize = text_size;
	int treeButtomSize = 512;
	//上層
	for (int i = 1; i < 10; i++)
	{	
		int * treeArray_pre_gpu; 
		int * treeArray_now_gpu;
		hipMalloc((void**)& treeArray_pre_gpu, sizeof(int)* totalSize);
		hipMalloc((void**)& treeArray_now_gpu, sizeof(int)* totalSize/2);
		hipMemcpy(treeArray_pre_gpu, treeLayer[i - 1], sizeof(int)*totalSize, hipMemcpyHostToDevice);
		
		block_dim = (totalSize) / treeButtomSize + 1;
		EstablishLayerTree << <block_dim, treeButtomSize >> >(treeArray_pre_gpu, treeArray_now_gpu, totalSize);
		hipDeviceSynchronize();

		hipMemcpy(treeLayer[i], treeArray_now_gpu, sizeof(int)*totalSize / 2, hipMemcpyDeviceToHost);
		if (totalSize == 1) break;
		treeButtomSize /= 2;
		totalSize /= 2;
	}
	//全塞到大ARRAY用一維陣列方式傳遞
	int * treeBigArray_cpu = (int *)malloc(sizeof(int)* text_size * 2);
	int * treeBigArray_gpu;
	hipMalloc((void**)&treeBigArray_gpu, sizeof(int)*text_size * 2);
	
	int all_Idx=0;
	for (int i = 0, size = text_size; i<10; i++,size/=2)
	{
		int *treeTmp_cpu = (int *)malloc(sizeof(int)*size);
		hipMemcpy(treeTmp_cpu, treeLayer[i], sizeof(int)*size, hipMemcpyHostToHost);
		for (int j = 0; j<size; j++)
		{
			treeBigArray_cpu[all_Idx] = treeTmp_cpu[j];
			all_Idx++;
			printf("%d ", treeTmp_cpu[j]);
		}
		printf("\n");
	}
	hipMemcpy(treeBigArray_gpu, treeBigArray_cpu, sizeof(int)* text_size * 2, hipMemcpyHostToDevice);

	block_dim = text_size / 512 + 1;
	SetPosition << < text_size, 512 >> >(treeBigArray_gpu, pos, text_size);
	hipDeviceSynchronize();
	/*
	int  *treeLayerBeginSize[10];
	int** treeArray;
	treeArray = (int **)malloc(10 * sizeof(int *));
	int size = text_size;
	for (int i = 0; i< 10; i++){
		treeArray[i] = (int *)malloc(sizeof(int)*size);
		treeLayerBeginSize[i] = treeArray[i];
		hipMalloc((void**)&treeLayerBeginSize[i], sizeof(int)*size);
		hipMemcpy(treeLayerBeginSize[i], treeArray[i], sizeof(int)*text_size, hipMemcpyHostToDevice);
		size = size / 2;
	}
	int block_dim = text_size / 512+1;

	EstablishButtonTree << <block_dim, 512 >> >(text, treeLayerBeginSize[0], text_size);
	int totalSize = text_size;
	int treeButtomSize = 256;
	hipDeviceSynchronize();
	for (int i = 1; i < 10; i++)
	{
		block_dim = (totalSize) / treeButtomSize+1;
		EstablishLayerTree << <block_dim, treeButtomSize >> >(treeLayerBeginSize[i - 1], treeLayerBeginSize[i], totalSize);
		hipDeviceSynchronize();
		treeButtomSize /= 2;
		totalSize =  totalSize / 2;

	}
	 block_dim = text_size / 512+1;
	SetPosition << < text_size, 1 >> >(treeLayerBeginSize, text_size);
	//SetCount << <block_dim, 512 >> >(treeLayerBeginSize[0], text_size);
	hipMemcpy(pos, treeLayerBeginSize[0], sizeof(int)*text_size, hipMemcpyDeviceToHost);*/
}
template<int N>
class compare {
public:
	__device__ bool operator () (int x) { return x == N; }
};

int ExtractHead(const int *pos, int *head, int text_size)
{
	int *buffer;
	int nhead = 0;
	hipMalloc((void**)&buffer, sizeof(int)*text_size * 2); // this is enough
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head), flag_d(buffer), cumsum_d(buffer + text_size);
	
	// TODO 如果pos_d是1就把thrust::counting_iterator<int>(text_size)內容存入head_d
	auto head_end_d =
	thrust::copy_if(
	thrust::counting_iterator<int>(0),
	thrust::counting_iterator<int>(text_size),
	pos_d,
	head_d,
	compare<1>()
	);
	nhead = head_end_d - head_d;
	hipFree(buffer);
	return nhead;
}
__global__ void SomeTransform(char *text, int *pos, int textSize, int *head, int n_head) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < textSize ) {
		for (int i = 0; i<n_head;i++){//把每個頭設定7
		if (idx == head[i])pos[idx] = 7;
		}
	}
}
void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{
	int blockNum = text_size / 32+1;
	SomeTransform << <blockNum, 32 >> >(text, pos, text_size, head, n_head);
}
